#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdbool.h>

#define SHA256_BLOCK_SIZE 32
#define MAX_NONCE 0xFFFFFFFF

// Define missing macros
#define ROTRIGHT(word, bits) (((word) >> (bits)) | ((word) << (32 - (bits))))
#define CH(x, y, z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))

// SHA-256 constants
__device__ __constant__ uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
    0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
    0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
    0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
    0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

// Atomic flag to signal found nonce
__device__ bool found = false;

// Device-compatible strlen function
__device__ size_t device_strlen(const char* str) {
    size_t len = 0;
    while (str[len] != '\0') {
        len++;
    }
    return len;
}

__device__ void sha256_transform(uint32_t state[8], const uint8_t data[64]) {
    uint32_t a, b, c, d, e, f, g, h, t1, t2, m[64];

    // Parse first 16 words
    for (int i = 0, j = 0; i < 16; ++i, j += 4)
        m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);

    // Extend the first 16 words into the remaining 48 words
    for (int i = 16; i < 64; ++i)
        m[i] = ROTRIGHT(m[i - 15], 7) ^ ROTRIGHT(m[i - 15], 18) ^ (m[i - 15] >> 3) +
               m[i - 7] + ROTRIGHT(m[i - 2], 17) ^ ROTRIGHT(m[i - 2], 19) ^ (m[i - 2] >> 10) +
               m[i - 16];

    a = state[0];
    b = state[1];
    c = state[2];
    d = state[3];
    e = state[4];
    f = state[5];
    g = state[6];
    h = state[7];

    for (int i = 0; i < 64; ++i) {
        t1 = h + (ROTRIGHT(e, 6) ^ ROTRIGHT(e, 11) ^ ROTRIGHT(e, 25)) +
             CH(e, f, g) + k[i] + m[i];
        t2 = (ROTRIGHT(a, 2) ^ ROTRIGHT(a, 13) ^ ROTRIGHT(a, 22)) +
             MAJ(a, b, c);
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    state[5] += f;
    state[6] += g;
    state[7] += h;
}

__device__ void sha256_init(uint32_t state[8]) {
    state[0] = 0x6a09e667;
    state[1] = 0xbb67ae85;
    state[2] = 0x3c6ef372;
    state[3] = 0xa54ff53a;
    state[4] = 0x510e527f;
    state[5] = 0x9b05688c;
    state[6] = 0x1f83d9ab;
    state[7] = 0x5be0cd19;
}

__device__ void sha256_final(uint32_t state[8], uint8_t hash[32]) {
    for (int i = 0; i < 8; ++i) {
        hash[i * 4]     = (state[i] >> 24) & 0xff;
        hash[i * 4 + 1] = (state[i] >> 16) & 0xff;
        hash[i * 4 + 2] = (state[i] >> 8) & 0xff;
        hash[i * 4 + 3] = state[i] & 0xff;
    }
}

__device__ void sha256_compute(uint32_t state[8], const uint8_t data[], size_t len) {
    size_t i;
    uint8_t block[64];
    size_t processed = 0;

    while (processed + 64 <= len) {
        memcpy(block, data + processed, 64);
        sha256_transform(state, block);
        processed += 64;
    }

    // Handle remaining data and padding (simplified for nonce hashing)
    memset(block, 0, 64);
    memcpy(block, data + processed, len - processed);
    // Primitively handle padding (not complete)
    sha256_transform(state, block);
}

__global__ void mineBlockKernel(const char* input, size_t input_len, uint32_t* nonce, uint8_t* hash, uint32_t difficulty) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t step = gridDim.x * blockDim.x;
    uint32_t localNonce = idx;

    uint32_t state[8];
    uint8_t localHash[32];

    // Initialize SHA-256 state
    sha256_init(state);

    // Precompute the static part of the data (input)
    // Assuming input is already properly formatted for hashing
    // Append nonce to input
    uint8_t data[64];
    memset(data, 0, 64);
    size_t copy_len = input_len < 64 ? input_len : 64;
    memcpy(data, input, copy_len);

    while (!found && localNonce < MAX_NONCE) {
        // Copy nonce into data
        memcpy(data + copy_len, &localNonce, sizeof(localNonce));

        // Compute SHA-256
        uint32_t temp_state[8];
        memcpy(temp_state, state, sizeof(uint32_t) * 8);
        sha256_compute(temp_state, data, copy_len + sizeof(localNonce));
        sha256_final(temp_state, localHash);

        // Check difficulty
        bool isValid = true;
        for (uint32_t i = 0; i < difficulty; ++i) {
            if (localHash[i] != 0) {
                isValid = false;
                break;
            }
        }

        if (isValid) {
            if (!atomicExch((int*)&found, 1)) { // Set found to true atomically
                *nonce = localNonce;
                memcpy(hash, localHash, SHA256_BLOCK_SIZE);
            }
            break;
        }

        localNonce += step;
    }
}

extern "C" bool mineBlock(const char* input, size_t input_len, uint8_t* output_hash, uint32_t* output_nonce, uint32_t difficulty) {
    char* d_input;
    uint8_t* d_hash;
    uint32_t* d_nonce;

    // Allocate device memory
    hipError_t err;
    err = hipMalloc((void**)&d_input, input_len);
    if (err != hipSuccess) {
        printf("CUDA malloc failed for input: %s\n", hipGetErrorString(err));
        return false;
    }

    err = hipMalloc((void**)&d_hash, SHA256_BLOCK_SIZE);
    if (err != hipSuccess) {
        printf("CUDA malloc failed for hash: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        return false;
    }

    err = hipMalloc((void**)&d_nonce, sizeof(uint32_t));
    if (err != hipSuccess) {
        printf("CUDA malloc failed for nonce: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_hash);
        return false;
    }

    // Initialize 'found' flag to false
    bool h_found = false;
    err = hipMemcpyToSymbol(HIP_SYMBOL(found), &h_found, sizeof(bool));
    if (err != hipSuccess) {
        printf("CUDA memcpy to symbol failed: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_hash);
        hipFree(d_nonce);
        return false;
    }

    // Copy input data to device
    err = hipMemcpy(d_input, input, input_len, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA memcpy failed for input: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_hash);
        hipFree(d_nonce);
        return false;
    }

    // Kernel launch configuration
    int blockSize = 256;
    int numBlocks = 256; // Adjust based on GPU's capability

    // Launch the mining kernel
    mineBlockKernel<<<numBlocks, blockSize>>>(d_input, input_len, d_nonce, d_hash, difficulty);

    // Wait for GPU to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA kernel failed: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_hash);
        hipFree(d_nonce);
        return false;
    }

    // Check if a nonce was found
    err = hipMemcpy(output_nonce, d_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("CUDA memcpy failed for nonce: %s\n", hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_hash);
        hipFree(d_nonce);
        return false;
    }

    // If a nonce was found, copy the hash
    if (*output_nonce != 0) {
        err = hipMemcpy(output_hash, d_hash, SHA256_BLOCK_SIZE, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            printf("CUDA memcpy failed for hash: %s\n", hipGetErrorString(err));
            hipFree(d_input);
            hipFree(d_hash);
            hipFree(d_nonce);
            return false;
        }
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_hash);
    hipFree(d_nonce);

    // Return whether a valid nonce was found
    return (*output_nonce != 0);
}